#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>

#define checkCuda(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline hipError_t cudaAssert(hipError_t result, const char *file, int line, bool abort = true) {
    if (result != hipSuccess){
        fprintf(stderr, "CUDA Error: \"%s\" in %s:%d\n", hipGetErrorString(result), file, line);
        if (abort) {
            exit(result);
        }
    }
    return result;
}

const int matrixWidth = 10;
const int matrixLength = 32;
const int blocksize = 16;

__global__ void revert(float *inputMatrix, float *resultMatrix, int maxYPosition, int maxXPosition)
{
    int columnNo = blockIdx.x * blockDim.x + threadIdx.x;
    int rowNo = blockIdx.y * blockDim.y + threadIdx.y;
    int index = rowNo * matrixWidth + columnNo;
    int revertIndex = maxXPosition * maxYPosition - 1 - index;
    if (columnNo < maxXPosition && rowNo < maxYPosition)
        resultMatrix[index] = inputMatrix[revertIndex];
}

int main() {
    // host declarations
    float *resultMatrix = new float[matrixWidth*matrixLength];
    float *inputMatrix = new float[matrixWidth*matrixLength];
    for (int i = 0; i < matrixLength * matrixWidth; ++i) {
        inputMatrix[i] = 2* i;
    }

    // cuda declarations
    const int matrixByteSize = matrixWidth * matrixLength * sizeof(float);
    float *c_resultMatrix, *c_inputMatrix;
    checkCuda(hipMalloc((void**)&c_resultMatrix, matrixByteSize));
    checkCuda(hipMalloc((void**)&c_inputMatrix, matrixByteSize));

    // copy host -> cuda
    checkCuda(hipMemcpy(c_inputMatrix, inputMatrix, matrixByteSize, hipMemcpyHostToDevice));

    // declare block and grid
    dim3 dimBlock(blocksize, blocksize);
    int block_columns_count = ceil((float) matrixWidth / (float) dimBlock.y);
    int block_row_count = ceil((float) matrixLength / (float) dimBlock.x);
    printf("grid dim: %d blocks height x %d blocks width\n", block_row_count, block_columns_count);
    dim3 dimGrid(block_columns_count, block_row_count);

    // call cuda function
    revert << <dimGrid, dimBlock >> > (c_inputMatrix, c_resultMatrix, matrixLength, matrixWidth);

    // copy cuda -> host
    checkCuda(hipMemcpy(resultMatrix, c_resultMatrix, matrixByteSize, hipMemcpyDeviceToHost));

    checkCuda(hipDeviceSynchronize());
    // display
    for (int row = 0; row < matrixLength; row++) {
        for (int column=0; column < matrixWidth; column++) {
            printf("%d ", (int) resultMatrix[row * matrixWidth + column]);
        }
        printf("\n");
    }
	for (int row = 0; row < matrixLength * matrixWidth; row++) {
		printf("%d ", (int) resultMatrix[row]);
	}

    // free memory on cuda and host
    hipFree(c_resultMatrix);
    hipFree(c_inputMatrix);
    delete[] resultMatrix;
    delete[] inputMatrix;
    return 0;
}

