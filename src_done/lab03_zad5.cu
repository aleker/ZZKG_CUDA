#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <iostream>     // cout
#include <algorithm>

#define checkCuda(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline hipError_t cudaAssert(hipError_t result, const char *file, int line, bool abort = true) {
    if (result != hipSuccess){
        fprintf(stderr, "CUDA Error: \"%s\" in %s:%d\n", hipGetErrorString(result), file, line);
        if (abort) {
            exit(result);
        }
    }
    return result;
}

const int blocksize = 32;
const int blockCount = 2;
const int n = blocksize * blockCount;    // vector size
const int sharedArraySize = n;


__global__ void sum_front(int *inputVector, float *resultVector, int inputFullSize) {
    extern __shared__ float shared[];

    // 1 block per row
    // (blocksize x blockCount matrix)
    // inputFullSize = blocksize * blockCount
    int rowLength = blockDim.x;
    int columnNo = blockIdx.x * blockDim.x + threadIdx.x;
    int rowNo = blockIdx.y * blockDim.y + threadIdx.y;

    int tid = rowNo * rowLength + columnNo; // global thread id (gtid)
    int bid = threadIdx.x;      // thread id in block (btid)

    // write to shared
    // bid == tid
    if (tid < inputFullSize) {
        shared[bid] = inputVector[tid];
    }
    __syncthreads();

    // write result
    if (tid < inputFullSize) {
        int sum = 0;
        for (int i = 0; i <= bid; i++) {
            sum += shared[i];
        }
        resultVector[tid] = sum;
    }
    __syncthreads();
}

int main() {
    thrust::host_vector<int> h_tab;
    thrust::device_vector<int> d_tab;

    thrust::host_vector<float> h_tab_result;
    thrust::device_vector<float> d_tab_result;

    // host initialization
    for (int i=0; i < n; i++) {
        h_tab.push_back(2*i);
        std::cout << h_tab[i] << " ";
    }
    std::cout << "\n";

    //Kopiowanie host->device
    d_tab = h_tab;
    d_tab_result.resize(n);    // zajmuje i uznaje za wypelnione

    sum_front << <blockCount, blocksize, sharedArraySize * blockCount * sizeof(int)>> > (d_tab.data().get(), d_tab_result.data().get(), d_tab.size());

    //Kopiowanie device->host
    h_tab_result = d_tab_result;

    for (int i = 0; i < n; i++) {
        std::cout << h_tab_result[i] << " ";
    }

    return 0;
}

