#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <iostream>     // cout
#include <algorithm>

#define checkCuda(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline hipError_t cudaAssert(hipError_t result, const char *file, int line, bool abort = true) {
    if (result != hipSuccess){
        fprintf(stderr, "CUDA Error: \"%s\" in %s:%d\n", hipGetErrorString(result), file, line);
        if (abort) {
            exit(result);
        }
    }
    return result;
}

const int blocksize = 32;
const int blockCount = 4;
const int n = blocksize * blockCount;    // vector size
const int sharedArraySize = n;


__global__ void sum_front(int *inputVector, int *resultVector, int inputFullSize) {
    extern __shared__ float shared[];

    // 1 block per row
    // (blocksize x blockCount matrix)
    // inputFullSize = blocksize * blockCount
    int rowLength = blockDim.x;
    int columnNo = blockIdx.x * blockDim.x + threadIdx.x;
    int rowNo = blockIdx.y * blockDim.y + threadIdx.y;

    int tid = rowNo * rowLength + columnNo; // global thread id (gtid)
    int bid = threadIdx.x;      // thread id in block (btid)

    // write to shared
    // bid == tid
    if (tid < inputFullSize) {
        shared[bid] = inputVector[tid];
    }
    __syncthreads();

    // write result
    if (tid < inputFullSize) {
        int sum = 0;
        for (int i = 0; i <= bid; i++) {
            sum += shared[i];
        }
        resultVector[tid] = sum;
    }
    __syncthreads();
}

__global__ void sum_front_return_last(int *inputVector, int *resultVector, int inputFullSize) {
    extern __shared__ float shared[];

    // 1 block per row
    // (blocksize x blockCount matrix)
    // inputFullSize = blocksize * blockCount
    int rowLength = blockDim.x;
    int columnNo = blockIdx.x * blockDim.x + threadIdx.x;
    int rowNo = blockIdx.y * blockDim.y + threadIdx.y;

    int tid = rowNo * rowLength + columnNo; // global thread id (gtid)
    int bid = threadIdx.x;      // thread id in block (btid)

    // write to shared
    // bid == tid
    if (tid < inputFullSize) {
        shared[bid] = inputVector[tid];
    }
    __syncthreads();

    // write result
    if (tid < inputFullSize) {
        int sum = 0;
        for (int i = 0; i <= bid; i++) {
            sum += shared[i];
        }
        if (bid == blockDim.x - 1)
            resultVector[blockIdx.x] = sum;
    }
    __syncthreads();
}

int main() {
    thrust::host_vector<int> h_tab;
    thrust::device_vector<int> d_tab;

    thrust::host_vector<int> h_tab_result;
    thrust::device_vector<int> d_tab_result;

    thrust::host_vector<int> h_tab_result_first;
    thrust::device_vector<int> d_tab_result_first;

    // host initialization
    for (int i=1; i <= blockCount; i++) {
        for (int j=0; j < blocksize; j++) {
            h_tab.push_back(i*j);
            std::cout << h_tab[i*j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";

    //Kopiowanie host->device
    d_tab = h_tab;
    d_tab_result_first.resize(blockCount);    // zajmuje i uznaje za wypelnione

    sum_front_return_last << <blockCount, blocksize, sharedArraySize * blockCount * sizeof(int)>> > (d_tab.data().get(), d_tab_result_first.data().get(), d_tab.size());
    h_tab_result_first = d_tab_result_first;
    for (int i = 0; i < blockCount; i++) {
        std::cout << h_tab_result_first[i] << " ";
    }
    std::cout << "\n";

    sum_front << <1, blockCount, sharedArraySize * blockCount * sizeof(int)>> > (d_tab_result_first.data().get(), d_tab_result.data().get(), d_tab_result_first.size());

    //Kopiowanie device->host
    h_tab_result = d_tab_result;

    for (int i = 0; i < blockCount; i++) {
        std::cout << h_tab_result[i] << " ";
    }
    std::cout << "\n";

    return 0;
}

